#include "hip/hip_runtime.h"
#include <iostream>
#include <cstdio>
#include <cmath>

#include <opencv2/core/core.hpp>
#include <opencv2/highgui/highgui.hpp>

#include "../common/common.h"
#include <hip/hip_runtime.h>

using namespace std;


//Adrian Biller A10018940 gpu blur code


//convolution matrix dimension *must be odd number*
#define BLURMATRIX 5
//nvcc -o test Mult2D_1D.cu -std=c++11


// input - input image one dimensional array
// ouput - output image one dimensional array
// width, height - width and height of the images
// colorWidthStep - number of color bytes (cols * colors)
// grayWidthStep - number of gray bytes
__global__ void bgr_to_gray_kernel(unsigned char* input, unsigned char* output, int width, int height, int colorWidthStep, int grayWidthStep)
{
	// 2D Index of current thread
	const int xIndex = blockIdx.x * blockDim.x + threadIdx.x;
	const int yIndex = blockIdx.y * blockDim.y + threadIdx.y;

	// Only valid threads perform memory I/O
	if ((xIndex < width) && (yIndex < height))
	{
		int convolutionDims = BLURMATRIX * BLURMATRIX;
		int red = 0;
		int green = 0;
		int blue = 0;



		// cout<<" posicion ("<<i<<","<<j<<")"<<endl;
		//for that goes through the blur matrix area
		for(int convolutionX = xIndex - floor(BLURMATRIX/2); convolutionX < xIndex + floor(BLURMATRIX/2); convolutionX++){
			for(int convolutionY = yIndex - floor(BLURMATRIX/2); convolutionY < yIndex + floor(BLURMATRIX/2); convolutionY++){
				if(convolutionX > 0 && convolutionX < width && convolutionY > 0 && convolutionY < height){
					//getting every value of the BLURMATRIX
					blue += input.at<cv::Vec3b>(convolutionY, convolutionX)[0];
					green += input.at<cv::Vec3b>(convolutionY, convolutionX)[1];
					red += input.at<cv::Vec3b>(convolutionY, convolutionX)[2];
				}

			}
		}
		//mean of all pixels around the selected pixel
		blue = floor(blue/convolutionDims);
		green = floor(green/convolutionDims);
		red = floor(red/convolutionDims);
		// cout<<"("<<i<<","<<j<<")"<<endl;
		//asigning mean to target pixel
		output.at<cv::Vec3b>(j, i)[0] = blue;
		output.at<cv::Vec3b>(j, i)[1] = green;
		output.at<cv::Vec3b>(j, i)[2] = red;
	}
}

void blur_image(const cv::Mat& input, cv::Mat& output)
{
	cout << "Input image step: " << input.step << " rows: " << input.rows << " cols: " << input.cols << endl;
	// Calculate total number of bytes of input and output image
	// Step = cols * number of colors
	size_t colorBytes = input.step * input.rows;
	size_t blurBytes = output.step * output.rows;

	unsigned char *d_input, *d_output;

	// Allocate device memory
	SAFE_CALL(hipMalloc<unsigned char>(&d_input, colorBytes), "CUDA Malloc Failed");
	SAFE_CALL(hipMalloc<unsigned char>(&d_output, blurBytes), "CUDA Malloc Failed");

	// Copy data from OpenCV input image to device memory
	SAFE_CALL(hipMemcpy(d_input, input.ptr(), colorBytes, hipMemcpyHostToDevice), "CUDA Memcpy Host To Device Failed");

	// Specify a reasonable block size
	const dim3 block(32, 32);

	// Calculate grid size to cover the whole image
	// const dim3 grid((input.cols + block.x - 1) / block.x, (input.rows + block.y - 1) / block.y);
	const dim3 grid((int)ceil((float)input.cols / block.x), (int)ceil((float)input.rows/ block.y));
	printf("blur_kernel<<<(%d, %d) , (%d, %d)>>>\n", grid.x, grid.y, block.x, block.y);

	// Launch the color conversion kernel
	blur_kernel <<<grid, block >>>(d_input, d_output, input.cols, input.rows, static_cast<int>(input.step), static_cast<int>(output.step));

	// Synchronize to check for any kernel launch errors
	SAFE_CALL(hipDeviceSynchronize(), "Kernel Launch Failed");

	// Copy back data from destination device meory to OpenCV output image
	SAFE_CALL(hipMemcpy(output.ptr(), d_output, grayBytes, hipMemcpyDeviceToHost), "CUDA Memcpy Host To Device Failed");

	// Free the device memory
	SAFE_CALL(hipFree(d_input), "CUDA Free Failed");
	SAFE_CALL(hipFree(d_output), "CUDA Free Failed");
}

int main(int argc, char *argv[])
{
	string imagePath;

	if(argc < 2)
		imagePath = "small_img.jpg";
  	else
  		imagePath = argv[1];

	// Read input image from the disk
	cv::Mat input = cv::imread(imagePath, CV_LOAD_IMAGE_COLOR);

	if (input.empty())
	{
		cout << "Image Not Found!" << std::endl;
		cin.get();
		return -1;
	}

	//Create output image										Formato de imagen
	cv::Mat output(input.rows, input.cols, CV_8UC1);

	//Call the wrapper function
	convert_to_gray(input, output);

	//Allow the windows to resize
	namedWindow("Input", cv::WINDOW_NORMAL);
	namedWindow("Output", cv::WINDOW_NORMAL);

	//Show the input and output
	imshow("Input", input);
	imshow("Output", output);

	//Wait for key press
	cv::waitKey();

	return 0;
}
